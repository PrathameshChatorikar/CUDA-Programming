#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate thread index
    if (index < N) {
        C[index] = A[index] + B[index];  // Perform element-wise addition
    }
}

int main() {
    int N = 1 << 20;  // Number of elements in the vector (2^20 = 1048576 elements)
    size_t size = N * sizeof(float);  // Size of memory to allocate (in bytes)

    // Allocate memory for vectors on the host (CPU)
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize vectors A and B with sample values
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate memory for vectors on the device (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel for vector addition
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Check for any errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Copy the result vector back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (h_C[i] != 3.0f) {
            std::cerr << "Error: Element " << i << " is " << h_C[i] << " instead of 3.0" << std::endl;
            return -1;
        }
    }

    std::cout << "Vector addition successful!" << std::endl;

    // Free the allocated memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
