#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void count_nonzero(const int* input, int* count, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && input[idx] != 0) {
        atomicAdd(count, 1);
    }
}

__global__ void collect_nonzero_indices(const int* input, int* output, int* pos, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size && input[idx] != 0) {
        int out_idx = atomicAdd(pos, 1);
        output[out_idx] = idx;
    }
}

void cuda_argnonzero(const std::vector<int>& host_input, std::vector<int>& host_output) {
    int size = host_input.size();
    int *d_input, *d_output, *d_count;

    hipMalloc(&d_input, size * sizeof(int));
    hipMemcpy(d_input, host_input.data(), size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));

    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    // Count non-zero elements
    count_nonzero<<<blocks, threads>>>(d_input, d_count, size);
    hipDeviceSynchronize();

    int count;
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    hipMalloc(&d_output, count * sizeof(int));
    hipMemset(d_count, 0, sizeof(int));  // reuse d_count as position counter

    // Collect non-zero indices
    collect_nonzero_indices<<<blocks, threads>>>(d_input, d_output, d_count, size);
    hipDeviceSynchronize();

    host_output.resize(count);
    hipMemcpy(host_output.data(), d_output, count * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_count);
}

int main() {
    std::vector<int> input = {0, 4, 0, 5, 6, 0, 7};
    std::vector<int> output;

    cuda_argnonzero(input, output);

    std::cout << "Indices of non-zero elements: ";
    for (int idx : output) std::cout << idx << " ";
    std::cout << std::endl;

    return 0;
}
