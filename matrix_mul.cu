#include <iostream>
#include <hip/hip_runtime.h>

#define M 4
#define N 4
#define P 4

__global__ void matMulKernel(int *A, int *B, int *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < P) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * P + col];
        }
        C[row * P + col] = sum;
    }
}

int main() {
    int sizeA = M * N * sizeof(int);
    int sizeB = N * P * sizeof(int);
    int sizeC = M * P * sizeof(int);

    int h_A[M * N], h_B[N * P], h_C[M * P];

    // Fill A and B with sample values
    for (int i = 0; i < M * N; i++) h_A[i] = 1;
    for (int i = 0; i < N * P; i++) h_B[i] = 2;

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((P + 15) / 16, (M + 15) / 16);
    matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Matrix C (Result):" << std::endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++) {
            std::cout << h_C[i * P + j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}
